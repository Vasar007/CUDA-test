#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>

#include "utils.hpp"


__global__ void kernel(int* const c, const int* const a, const int* const b, const int size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}


void print_short(const int* const a, const int size, const int border)
{
    for (int i = 0; i < border; ++i)
    {
        std::cout << a[i] << '\t';
    }
    std::cout << "..\t";
    for (int i = size - border; i < size; ++i)
    {
        std::cout << a[i] << '\t';
    }
    std::cout << '\n';
}


int main()
{
    constexpr int lower_bound = 1;
    constexpr int top_bound = 10;

    constexpr int size = 512;
    int a[size];
    int b[size];
    int c[size];
    for (int i = 0; i < size; ++i)
    {
        a[i] = utils::random_number(lower_bound, top_bound);
        b[i] = utils::random_number(lower_bound, top_bound);
    }
    print_short(a, size, 5);
    print_short(b, size, 5);

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;
    hipMalloc((void**) &dev_a, size * sizeof(int));
    hipMalloc((void**) &dev_b, size * sizeof(int));
    hipMalloc((void**) &dev_c, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float worktime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    const int blocks_opts[] = { 1, 2, 4, 8, 16, 32, 64, 128, 256, 512 };

    for (int i = 0; i < 10; ++i)
    {
        hipEventRecord(start, 0);
        kernel<<<blocks_opts[i], size / blocks_opts[i]>>>(dev_c, dev_a, dev_b, size);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&worktime, start, stop);
        std::cout << blocks_opts[i] << ", " << size / blocks_opts[i] << ":\t"
                  << std::fixed << std::setprecision(16) << worktime << '\n';
    }

    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    print_short(c, size, 5);

    return 0;
}

