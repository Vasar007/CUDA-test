#include "hip/hip_runtime.h"
#include <iostream>

#include "utils.hpp"


__global__ void kernel(int* const c, const int* const a, const int* const b, const int size)
{
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < size && iy < size)
    {
        const int offset = iy * size + ix;
        c[offset] = a[offset] + b[offset];
    }
}

void print_short(const int* const * const a, const int size, const int border)
{
    for (int i = 0; i < border; ++i)
    {
        for (int j = 0; j < border; ++j)
        {
            std::cout << a[i][j] << '\t';
        }
        std::cout << "..\t";
        for (int j = size - border; j < size; ++j)
        {
            std::cout << a[i][j] << '\t';
        }
        std::cout << '\n';
    }

    std::cout << "..\t\n";

    for (int i = size - border; i < size; ++i)
    {
        for (int j = 0; j < border; ++j)
        {
            std::cout << a[i][j] << '\t';
        }
        std::cout << "..\t";
        for (int j = size - border; j < size; ++j)
        {
            std::cout << a[i][j] << '\t';
        }
        std::cout << '\n';
    }
    std::cout << '\n';
}

int main()
{
    constexpr int lower_bound = 1;
    constexpr int top_bound = 10;

    constexpr int size = 4000;
    int **a = new int*[size];
    int **b = new int*[size];
    int **c = new int*[size];
    a[0] = new int[size * size];
    b[0] = new int[size * size];
    c[0] = new int[size * size];

    for (int i = 1; i < size; ++i)
    {
        a[i] = a[i - 1] + size;
        b[i] = b[i - 1] + size;
        c[i] = c[i - 1] + size;
    }
    for (int i = 0; i < size; ++i)
    {
        for (int j = 0; j < size; ++j)
        {
            a[i][j] = utils::random_number(lower_bound, top_bound);
            b[i][j] = utils::random_number(lower_bound, top_bound);
        }
    }

    std::cout << "===== A: =====\n";
    print_short(a, size, 3);
    std::cout << "===== B: =====\n";
    print_short(b, size, 3);

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;
    hipMalloc((void**) &dev_a, size * size * sizeof(int));
    hipMalloc((void**) &dev_b, size * size * sizeof(int));
    hipMalloc((void**) &dev_c, size * size * sizeof(int));

    hipMemcpy(dev_a, a[0], size * size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b[0], size * size * sizeof(int), hipMemcpyHostToDevice);

    constexpr int threads_per_block_dimension = 16;
    dim3 blocks(size / threads_per_block_dimension, size / threads_per_block_dimension);
    dim3 threads(threads_per_block_dimension, threads_per_block_dimension);
    kernel<<<blocks, threads>>>(dev_c, dev_a, dev_b, size);

    hipDeviceSynchronize();

    hipMemcpy(c[0], dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    std::cout << "===== C: =====\n";
    print_short(c, size, 3);

    delete[] a[0];
    delete[] b[0];
    delete[] c[0];
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

